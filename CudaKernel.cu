#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"

#define PI 3.1415926535897932384626433832795

__device__ float3 matMul(const float3 r012, const float3 r345, const float3 r678, float3 v){
	float3 outvec = { 0, 0, 0 };
	outvec.x = r012.x * v.x + r012.y * v.y + r012.z * v.z;
	outvec.y = r345.x * v.x + r345.y * v.y + r345.z * v.z;
	outvec.z = r678.x * v.x + r678.y * v.y + r678.z * v.z;
	return outvec;
}

__device__ float2 repairUv(float2 uv){
	float2 outuv;

	if(uv.x<0) {
		outuv.x = 1.0 + uv.x;
		}else if(uv.x > 1.0){
			outuv.x = uv.x -1.0;
		} else {
			outuv.x = uv.x;
		}

		if(uv.y<0) {
			outuv.y = 1.0 + uv.y;
		} else if(uv.y > 1.0){
			outuv.y = uv.y -1.0;
		} else {
			outuv.y = uv.y;
		}

	outuv.x = min(max(outuv.x, 0.0), 1.0);
	outuv.y = min(max(outuv.y, 0.0), 1.0);

	return outuv;
}

__device__ float2 polarCoord(float3 dir) {	
	float3 ndir = normalize(dir);
	float longi = -atan2(ndir.z, ndir.x);
	
	float lat = acos(-ndir.y);
	
	float2 uv;
	uv.x = longi;
	uv.y = lat;
	
	float2 pitwo = {PI, PI};
	uv /= pitwo;
	uv.x /= 2.0;
	float2 ones = {1.0, 1.0};
	uv = fmodf(uv, ones);
	return uv;
}


__device__ float3 fisheyeDir(float3 dir, const float3 r012, const float3 r345, const float3 r678) {

	if (dir.x == 0 && dir.y == 0)
		return matMul(r012, r345, r678, dir);
		
	dir.x = dir.x / dir.z;
	dir.y = dir.y / dir.z;
	dir.z = 1;
	
	float2 uv;
	uv.x = dir.x;
	uv.y = dir.y;
	float r = sqrtf(uv.x*uv.x + uv.y*uv.y);
	
	float phi = atan2f(uv.y, uv.x);
	
	float theta = r;
	
	float3 fedir = { 0, 0, 0 };
	fedir.x = sin(theta) * cos(phi);
	fedir.y = sin(theta) * sin(phi);
	fedir.z = cos(theta);

	fedir = matMul(r012, r345, r678, fedir);
	
	return fedir;
}

__device__ float3 tinyPlanetSph(float3 uv) {
	if (uv.x == 0 && uv.y == 0)
		return uv;

    float3 sph;
	float2 uvxy;
	uvxy.x = uv.x/uv.z;
	uvxy.y = uv.y/uv.z;

	float u  =length(uvxy);
	float alpha = atan2(2.0f, u);
	float phi = PI - 2*alpha;
	float z = cos(phi);
	float x = sin(phi);
	
	uvxy = normalize(uvxy);
	
	sph.z = z;
	
	float2 sphxy = uvxy * x;

	sph.x = sphxy.x;
	sph.y = sphxy.y;
	
	return sph;
}

__device__ float4 linInterpCol(float2 uv, const float* input, int width, int height){
	float4 outCol = {0,0,0,0};
	float i = floor(uv.x);
	float j = floor(uv.y);
	float a = uv.x-i;
	float b = uv.y-j;
	int x = (int)i;
	int y = (int)j;
	int x1 = (x < width - 1 ? x + 1 : x);
	int y1 = (y < height - 1 ? y + 1 : y);
	const int indexX1Y1 = ((y * width) + x) * 4;
	const int indexX2Y1 = ((y * width) + x1) * 4;
	const int indexX1Y2 = (((y1) * width) + x) * 4;
	const int indexX2Y2 = (((y1) * width) + x1) * 4;
	const int maxIndex = (width * height -1) * 4;
	
	if(indexX2Y2 < maxIndex){
		outCol.x = (1.0 - a)*(1.0 - b)*input[indexX1Y1] + a*(1.0 - b)*input[indexX2Y1] + (1.0 - a)*b*input[indexX1Y2] + a*b*input[indexX2Y2];
		outCol.y = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 1] + a*(1.0 - b)*input[indexX2Y1 + 1] + (1.0 - a)*b*input[indexX1Y2 + 1] + a*b*input[indexX2Y2 + 1];
		outCol.z = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 2] + a*(1.0 - b)*input[indexX2Y1 + 2] + (1.0 - a)*b*input[indexX1Y2 + 2] + a*b*input[indexX2Y2 + 2];
		outCol.w = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 3] + a*(1.0 - b)*input[indexX2Y1 + 3] + (1.0 - a)*b*input[indexX1Y2 + 3] + a*b*input[indexX2Y2 + 3];
	} else {
		outCol.x = input[indexX1Y1];
		outCol.y = input[indexX1Y1+ 1];
		outCol.z = input[indexX1Y1+ 2];
		outCol.w = input[indexX1Y1+ 3];
	}
	return outCol;
}

__global__ void GainAdjustKernel(int p_Width, int p_Height, float* p_Fov, float* p_Tinyplanet, float* p_Rectilinear,
								const float* p_Input, float* p_Output, const float* r, int samples, bool bilinear)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
		const int index = ((y * p_Width) + x) * 4;

		float4 accum_col = {0, 0, 0, 0};

		for(int i=0; i<samples; i++){
			float fov = p_Fov[i];

		   float2 uv = { (float)x / p_Width, (float)y / p_Height };
		   float aspect = (float)p_Width / (float)p_Height;

		   float3 dir = { 0, 0, 0 };
		   dir.x = (uv.x * 2) - 1;
		   dir.y = (uv.y * 2) - 1;
		   dir.y /= aspect;
		   dir.z = fov;

		   float3 tinyplanet = tinyPlanetSph(dir);
		   tinyplanet = normalize(tinyplanet);

		   const float3 r012 = {r[i*9+0], r[i*9+1], r[i*9+2]};
		   const float3 r345 = {r[i*9+3], r[i*9+4], r[i*9+5]};
		   const float3 r678 = {r[i*9+6], r[i*9+7], r[i*9+8]};

		   tinyplanet = matMul(r012, r345, r678, tinyplanet);
		   float3 rectdir = matMul(r012, r345, r678, dir);

		   rectdir = normalize(rectdir);
		   dir = lerp(fisheyeDir(dir, r012, r345, r678), tinyplanet, p_Tinyplanet[i]);
		   dir = lerp(dir, rectdir, p_Rectilinear[i]);

		   float2 iuv = polarCoord(dir);
		   iuv = repairUv(iuv);

		   int x_new = iuv.x * (p_Width - 1);
		   int y_new = iuv.y * (p_Height - 1);

		   iuv.x *= (p_Width - 1);
		   iuv.y *= (p_Height - 1);

		   if ((x_new < p_Width) && (y_new < p_Height))
		   {
			   const int index_new = ((y_new * p_Width) + x_new) * 4;

			   float4 interpCol;
			   if (bilinear){
				   interpCol = linInterpCol(iuv, p_Input, p_Width, p_Height);
			   }
			   else {
				   interpCol = { p_Input[index_new + 0], p_Input[index_new + 1], p_Input[index_new + 2], p_Input[index_new + 3] };
			   }

			   accum_col.x += interpCol.x;
			   accum_col.y += interpCol.y;
			   accum_col.z += interpCol.z;
			   accum_col.w += interpCol.w;
			}
		}
		p_Output[index + 0] = accum_col.x / samples;
		p_Output[index + 1] = accum_col.y / samples;
		p_Output[index + 2] = accum_col.z / samples;
		p_Output[index + 3] = accum_col.w / samples;
   }
}

void RunCudaKernel(int p_Width, int p_Height, float* p_Fov, float* p_Tinyplanet, float* p_Rectilinear, const float* p_Input, float* p_Output, const float* p_RotMat, int p_Samples, bool p_Bilinear)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

	float* dev_rmat;
	hipMalloc((void**)&dev_rmat, sizeof(float)*9*p_Samples);
	hipMemcpy((void*)dev_rmat, (void*)p_RotMat, sizeof(float)*9*p_Samples, hipMemcpyHostToDevice);

	float* dev_fov;
	hipMalloc((void**)&dev_fov, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_fov, (void*)p_Fov, sizeof(float)*p_Samples, hipMemcpyHostToDevice);

	float* dev_tinyplanet;
	hipMalloc((void**)&dev_tinyplanet, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_tinyplanet, (void*)p_Tinyplanet, sizeof(float)*p_Samples, hipMemcpyHostToDevice);
	
	float* dev_rectilinear;
	hipMalloc((void**)&dev_rectilinear, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_rectilinear, (void*)p_Rectilinear, sizeof(float)*p_Samples, hipMemcpyHostToDevice);


    GainAdjustKernel<<<blocks, threads>>>(p_Width, p_Height, dev_fov, dev_tinyplanet, dev_rectilinear,
											p_Input, p_Output, dev_rmat, p_Samples, p_Bilinear);
	hipFree( dev_rmat );
	hipFree( dev_fov );
	hipFree( dev_tinyplanet );
	hipFree( dev_rectilinear );
}
